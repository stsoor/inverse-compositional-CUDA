#include "hip/hip_runtime.h"
#include <hip/hip_vector_types.h>
#include <helper_functions.h>
#include <iostream>
#include <opencv2/core.hpp>
#include <opencv2/opencv.hpp>

#include <vector>


#define HUBER_LOSS 0.80 //Huber loss function parameter, to redunce the influence of outliers
#define sobelKernelSize 3

cv::Mat buildImageMat(float* intensityArray, const std::size_t height, const std::size_t width)
{
    cv::Mat_<float> image(static_cast<int>(height), static_cast<int>(width));
    
    for(std::size_t row = 0; row != height; ++row)
    {
        for(std::size_t col = 0; col != width; ++col)
        {
             image.at<float>(static_cast<int>(row), static_cast<int>(col)) = intensityArray[row * width + col];
        }   
    }
    
    return image;
}

void buildTransformation(cv::Mat& W, float* p)
{
    W.at<float>(0,0) = p[0];
    W.at<float>(0,1) = p[2];
    W.at<float>(0,2) = p[4];
    W.at<float>(1,0) = p[1];
    W.at<float>(1,1) = p[3];
    W.at<float>(1,2) = p[5];
    W.at<float>(2,0) = 0.0f;
    W.at<float>(2,1) = 0.0f;
    W.at<float>(2,2) = 1.0f;
}

void init_warp(cv::Mat& W, cv::Mat p)
{
	W.at<float>(0, 0) = 1.0 + p.at<float>(0,0);
	W.at<float>(0, 1) = p.at<float>(2,0);
	W.at<float>(0, 2) = p.at<float>(4,0);
    
	W.at<float>(1, 0) = p.at<float>(1,0);
	W.at<float>(1, 1) = 1.0 + p.at<float>(3,0);
	W.at<float>(1, 2) = p.at<float>(5,0);
    
	W.at<float>(2, 0) = 0.0;
	W.at<float>(2, 1) = 0.0;
	W.at<float>(2, 2) = 1.0;
}

void update_warp(cv::Mat& W, cv::Mat idW)
{
    cv::Mat p =  (cv::Mat_<float>(6,1) <<   W.at<float>(0,0) - 1.0,   W.at<float>(1,0),   W.at<float>(0,1),   W.at<float>(1,1) - 1.0,   W.at<float>(0,2),   W.at<float>(1,2) );
    cv::Mat dp = (cv::Mat_<float>(6,1) << idW.at<float>(0,0) - 1.0, idW.at<float>(1,0), idW.at<float>(0,1), idW.at<float>(1,1) - 1.0, idW.at<float>(0,2), idW.at<float>(1,2) );
    
    //double det = 1 /((1 + dp[0]) * (1 + dp[3]) − dp[1] * dp[2]);
    //
    //if(fabs(det - 0) < 1e8)
    //{
    //    std::cout << "Degenerate warp, exiting." << std::endl;
    //    exit(1);
    //}
    
    cv::Mat_<float>newP(6,1);
    
    newP.at<float>(0,0) = p.at<float>(0,0) + dp.at<float>(0,0) + p.at<float>(0,0) * dp.at<float>(0,0) + p.at<float>(2,0) * dp.at<float>(1,0);
    newP.at<float>(1,0) = p.at<float>(1,0) + dp.at<float>(1,0) + p.at<float>(1,0) * dp.at<float>(0,0) + p.at<float>(3,0) * dp.at<float>(1,0);
    newP.at<float>(2,0) = p.at<float>(2,0) + dp.at<float>(2,0) + p.at<float>(0,0) * dp.at<float>(2,0) + p.at<float>(2,0) * dp.at<float>(3,0);
    newP.at<float>(3,0) = p.at<float>(3,0) + dp.at<float>(3,0) + p.at<float>(1,0) * dp.at<float>(2,0) + p.at<float>(3,0) * dp.at<float>(3,0);
    newP.at<float>(4,0) = p.at<float>(4,0) + dp.at<float>(4,0) + p.at<float>(0,0) * dp.at<float>(4,0) + p.at<float>(2,0) * dp.at<float>(5,0);
    newP.at<float>(5,0) = p.at<float>(5,0) + dp.at<float>(5,0) + p.at<float>(1,0) * dp.at<float>(4,0) + p.at<float>(3,0) * dp.at<float>(5,0);
    
    init_warp(W, newP);
}

template <class T>
T interpolate(cv::Mat& image, float y, float x)
{
  float xd, yd;  
  float k1 = modff(x,&xd);
  float k2 = modff(y,&yd);
  int xi = int(xd);
  int yi = int(yd);

  int f1 = xi < image.rows-1;  // Check that pixels to the right  
  int f2 = yi < image.cols-1; // and to down direction exist.

  T px1 = image.at<T>(yi  , xi);
  T px2 = image.at<T>(yi  , xi+1);
  T px3 = image.at<T>(yi+1, xi);
  T px4 = image.at<T>(yi+1, xi+1);      
  
  // Interpolate pixel intensity.
  T interpolated_value = 
  (1.0-k1)*(1.0-k2)*px1 +
  (f1 ? ( k1*(1.0-k2)*px2 ):0) +
  (f2 ? ( (1.0-k1)*k2*px3 ):0) +            
  ((f1 && f2) ? ( k1*k2*px4 ):0);

  return interpolated_value;
}

float norm(cv::Mat m)
{
    float squareSum = 0.0f;
    
    for(std::size_t row = 0; row < m.rows; ++row)
    {
        for(std::size_t col = 0; col < m.cols; ++col)
        {
            float elem = m.at<float>(static_cast<int>(row), static_cast<int>(col));
            
            squareSum += elem * elem;
        }
    }
    
    return sqrt(squareSum);
}

extern "C"
void inverseCompositional( float* imageArray
                         , float* templateImageArray
                         , float* affineParameterEstimates
                         , const std::size_t imageHeight
                         , const std::size_t imageWidth
                         , const std::size_t templateImageHeight
                         , const std::size_t templateImageWidth
                         , const float epsilon
                         , const int maxIteration
                         )
{
    cv::Mat target = buildImageMat(imageArray, imageHeight, imageWidth);
    cv::Mat source = buildImageMat(templateImageArray, templateImageHeight, templateImageWidth);
    
	// Find the 2-D similarity transform that best aligns the two images (uniform scale, rotation and translation)
	cv::Mat debug;

	cv::Mat source_gradient_row;    // Gradient of I in X direction.
	cv::Mat source_gradient_col;    // Gradient of I in Y direction.

									// Here we will store matrices.
	cv::Mat_<float> W(3,3);         // Current value of warp W(x,p)
	cv::Mat_<float> dW(3,3);        // Warp update.
	cv::Mat_<float> idW(3,3);       // Warp update.
	cv::Mat_<float> X(3,1);         // Point in coordinate frame of source.
	cv::Mat_<float> Z(3,1);         // Point in coordinate frame of target.

	cv::Mat_<float> H(6,6);         // Approximate Hessian.
	cv::Mat_<float> b(6,1);         // Vector in the right side of the system of linear equations.
	cv::Mat_<float> delta_p(6,1);   // Parameter update value.

							  // Create images.
	source_gradient_row = cv::Mat(source.rows, source.cols, CV_32FC1);
	source_gradient_col = cv::Mat(source.rows, source.cols, CV_32FC1);
    
    float* steepest_descent = new float[6 * templateImageHeight * templateImageWidth];

	//The "magic number" appearing at the end in the following is simply the inverse 
	//of the absolute sum of the weights in the matrix representing the Scharr filter.
	cv::Scharr(source, source_gradient_row, -1, 0, 1, 1.0 / 32.0);
	cv::Scharr(source, source_gradient_col, -1, 1, 0, 1.0 / 32.0);
    
	H = cv::Mat::zeros(6, 6, CV_32FC1);    
    
	int u, v;	// (u,v) - pixel coordinates in the coordinate frame of T.
	float u2, v2; // (u2,v2) - pixel coordinates in the coordinate frame of I.
	
	// Walk through pixels in the template T.
	int i, j;
	for(i=0; i< templateImageWidth; i++)
	{
		u = i;

		for(j=0; j < templateImageHeight; j++)
		{
			v = j;

			// Evaluate gradient of T.
			float Tx = source_gradient_col.at<float>(v, u);	
			float Ty = source_gradient_row.at<float>(v, u);	
			
			// Calculate steepest descent image's element.
            steepest_descent[6 * (v * templateImageWidth + u) + 0] = Tx * u;
            steepest_descent[6 * (v * templateImageWidth + u) + 1] = Ty * u;
            steepest_descent[6 * (v * templateImageWidth + u) + 2] = Tx * v;
            steepest_descent[6 * (v * templateImageWidth + u) + 3] = Ty * v;
            steepest_descent[6 * (v * templateImageWidth + u) + 4] = Tx;
            steepest_descent[6 * (v * templateImageWidth + u) + 5] = Ty;
            
			// Add a term to Hessian.
			int l,m;
			for(l=0;l<6;l++)
			{
				for(m=0;m<6;m++)
				{
                    H.at<float>(l, m) += steepest_descent[6 * (v * templateImageWidth + u) + l] * steepest_descent[6 * (v * templateImageWidth + u) + m];
				}
			}
		}
	}
    
    // Invert Hessian.
    cv::Mat iH = H.inv();

	/*
	 *   Iteration stage.
	 */
    
    buildTransformation(W, affineParameterEstimates);
    
    //cv::Mat R = (cv::Mat_<float>(3,3) << sqrt(2.0)/2.0, -sqrt(2.0)/2.0, 0, sqrt(2.0)/2.0, sqrt(2.0)/2.0, 0, 0, 0, 1);
    //W.at<float>(0,2) += 100;
    //W.at<float>(1,2) -= 100;
    
    cv::Mat R = (cv::Mat_<float>(3,3) << sqrt(3.0)/2, -1.0/2.0, 0, 1.0/2.0, sqrt(3.0)/2.0, 0, 0, 0, 1);
    W.at<float>(0,2) += 50;
    W.at<float>(1,2) -= 50;
    
    W = W * R;
    
    //W.at<float>(0,0) += 0.03;
    //W.at<float>(1,1) += 0.06;
    //W.at<float>(0,2) += 2;
    //W.at<float>(1,2) -= 3;
    

	// Here we will store current value of mean error.
	float mean_error=0;

	// Iterate
	int iter=0; // number of current iteration
    while(iter < maxIteration)
	{
		iter++; // Increment iteration counter

        target.copyTo(debug);
        
		mean_error = 0; // Set mean error value with zero

		int pixel_count=0; // Count of processed pixels
		
		b = cv::Mat::zeros(6, 1, CV_32FC1); // Set b matrix with zeroes
			
		// Walk through pixels in the template T.
		int i, j;
		for(i=0; i<templateImageWidth; i++)
		{
			int u = i;

			for(j=0; j< templateImageHeight; j++)
			{
				int v = j;

				// Set vector X with pixel coordinates (u,v,1)
                X = (cv::Mat_<float>(3,1) << u, v, 1);

				// Warp Z=W*X
                Z = W * X;

				// Get coordinates of warped pixel in coordinate frame of I.
                u2 = Z.at<float>(0,0);
                v2 = Z.at<float>(1,0);

				// Get the nearest integer pixel coords (u2i;v2i).
				int u2i = int(floor(u2));
				int v2i = int(floor(v2));

				if(u2i >= 0 && u2i < imageWidth && // check if pixel is inside I.
					v2i >= 0 && v2i < imageHeight)
				{
					pixel_count++;

					// Calculate intensity of a transformed pixel with sub-pixel accuracy
					// using bilinear interpolation.
					float I2 = interpolate<float>(target, v2, u2);
                    
                    debug.at<float>(v2i,u2i) = source.at<float>(v,u);
                    //if(v == 0 && u == 0 || v == 0 && u == source.cols-1 || v == source.rows-1 && u == 0 || v == source.rows-1 && u == source.cols-1)
                    if(v == 0 || u == 0 || u == source.cols-1 || v == source.rows-1)
                    {
                        debug.at<float>(v2i,u2i) = 1.0;
                    }

					// Calculate image difference D = I(W(x,p))-T(x).
					float D = I2 - source.at<float>(v, u);

					// Update mean error value.
					mean_error += fabs(D);

					// Add a term to b matrix.
					b.at<float>(0,0) += steepest_descent[6 * (v * templateImageWidth + u) + 0] * D;
					b.at<float>(1,0) += steepest_descent[6 * (v * templateImageWidth + u) + 1] * D;
					b.at<float>(2,0) += steepest_descent[6 * (v * templateImageWidth + u) + 2] * D;	
					b.at<float>(3,0) += steepest_descent[6 * (v * templateImageWidth + u) + 3] * D;	
					b.at<float>(4,0) += steepest_descent[6 * (v * templateImageWidth + u) + 4] * D;	
					b.at<float>(5,0) += steepest_descent[6 * (v * templateImageWidth + u) + 5] * D;					
				}	
			}
		}

		// Finally, calculate resulting mean error.
		if(pixel_count!=0)
			mean_error /= pixel_count;

		// Find parameter increment.
        delta_p = iH * b;

		init_warp(dW, delta_p);
		// Invert warp.
		idW = dW.inv();
        
        //W o idW;
		update_warp(W, idW);
        
        //dW.copyTo(W);

		// Print diagnostic information to screen.
		printf("iter=%d mean_error=%f\n", iter, mean_error);
        
        cv::imshow("Debug", debug);
        cv::waitKey(24);

		// Check termination critera.
		if(norm(delta_p) <= epsilon) break;
	}
    
    std::cout << W << std::endl;
    
    cv::imshow("Debug", debug);
    cv::waitKey(0);
}
