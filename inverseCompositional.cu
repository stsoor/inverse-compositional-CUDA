#include "hip/hip_runtime.h"
#include <hip/hip_vector_types.h>
#include <helper_functions.h>
#include <iostream>
#include <opencv2/core.hpp>
#include <opencv2/opencv.hpp>

#include <vector>


#define HUBER_LOSS 0.80 //Huber loss function parameter, to redunce the influence of outliers
#define sobelKernelSize 3

cv::Mat buildImageMat(float* intensityArray, const std::size_t height, const std::size_t width)
{
    cv::Mat_<float> image(static_cast<int>(height), static_cast<int>(width));
    
    for(std::size_t row = 0; row != height; ++row)
    {
        for(std::size_t col = 0; col != width; ++col)
        {
             image.at<float>(static_cast<int>(row), static_cast<int>(col)) = intensityArray[row * width + col];
        }   
    }
    
    return image;
}

void buildTransformationFromInput(cv::Mat& W, float* p)
{
    W.at<float>(0,0) = p[0];
    W.at<float>(0,1) = p[2];
    W.at<float>(0,2) = p[4];
    W.at<float>(1,0) = p[1];
    W.at<float>(1,1) = p[3];
    W.at<float>(1,2) = p[5];
    W.at<float>(2,0) = 0.0f;
    W.at<float>(2,1) = 0.0f;
    W.at<float>(2,2) = 1.0f;
}

void init_warp(cv::Mat& W, cv::Mat p)
{
	W.at<float>(0, 0) = 1.0 + p.at<float>(0,0);
	W.at<float>(0, 1) = p.at<float>(2,0);
	W.at<float>(0, 2) = p.at<float>(4,0);
    
	W.at<float>(1, 0) = p.at<float>(1,0);
	W.at<float>(1, 1) = 1.0 + p.at<float>(3,0);
	W.at<float>(1, 2) = p.at<float>(5,0);
    
	W.at<float>(2, 0) = 0.0;
	W.at<float>(2, 1) = 0.0;
	W.at<float>(2, 2) = 1.0;
}

cv::Mat getWarpInvert(cv::Mat W)
{
    float p1 = W.at<float>(0,0) - 1.0;
    float p2 = W.at<float>(1,0);
    float p3 = W.at<float>(0,1);
    float p4 = W.at<float>(1,1) - 1.0;
    float p5 = W.at<float>(0,2);
    float p6 = W.at<float>(1,2);
    
    float det = (1 + p1) * (1 + p4) - p2 * p3;
    
    cv::Mat invW = (cv::Mat_<float>(3,3) <<
                                            1.0 + (-p1 - p1 * p4 + p2 * p3) / det, (-p3) / det, (-p5 - p4 * p5 + p3 * p6) / det,
                                            (-p2) / det, 1.0 + (-p4 - p1 * p4 + p2 * p3) / det, (-p6 - p1 * p6 + p2 * p5) / det,
                                            0.0, 0.0, 1.0
                   );
    
    return invW;
}

void update_warp(cv::Mat& W, cv::Mat idW)
{
    cv::Mat p =  (cv::Mat_<float>(6,1) <<   W.at<float>(0,0) - 1.0,   W.at<float>(1,0),   W.at<float>(0,1),   W.at<float>(1,1) - 1.0,   W.at<float>(0,2),   W.at<float>(1,2) );
    cv::Mat dp = (cv::Mat_<float>(6,1) << idW.at<float>(0,0) - 1.0, idW.at<float>(1,0), idW.at<float>(0,1), idW.at<float>(1,1) - 1.0, idW.at<float>(0,2), idW.at<float>(1,2) );
        
    cv::Mat_<float>newP(6,1);
    
    newP.at<float>(0,0) = p.at<float>(0,0) + dp.at<float>(0,0) + p.at<float>(0,0) * dp.at<float>(0,0) + p.at<float>(2,0) * dp.at<float>(1,0);
    newP.at<float>(1,0) = p.at<float>(1,0) + dp.at<float>(1,0) + p.at<float>(1,0) * dp.at<float>(0,0) + p.at<float>(3,0) * dp.at<float>(1,0);
    newP.at<float>(2,0) = p.at<float>(2,0) + dp.at<float>(2,0) + p.at<float>(0,0) * dp.at<float>(2,0) + p.at<float>(2,0) * dp.at<float>(3,0);
    newP.at<float>(3,0) = p.at<float>(3,0) + dp.at<float>(3,0) + p.at<float>(1,0) * dp.at<float>(2,0) + p.at<float>(3,0) * dp.at<float>(3,0);
    newP.at<float>(4,0) = p.at<float>(4,0) + dp.at<float>(4,0) + p.at<float>(0,0) * dp.at<float>(4,0) + p.at<float>(2,0) * dp.at<float>(5,0);
    newP.at<float>(5,0) = p.at<float>(5,0) + dp.at<float>(5,0) + p.at<float>(1,0) * dp.at<float>(4,0) + p.at<float>(3,0) * dp.at<float>(5,0);
    
    init_warp(W, newP);
}

float interpolate(cv::Mat& image, float y, float x)
{
  float xd, yd;  
  float k1 = modff(x,&xd);
  float k2 = modff(y,&yd);
  int xi = int(xd);
  int yi = int(yd);

  int f1 = xi < image.rows-1;  // Check that pixels to the right  
  int f2 = yi < image.cols-1; // and to down direction exist.

  float px1 = image.at<float>(yi  , xi);
  float px2 = image.at<float>(yi  , xi+1);
  float px3 = image.at<float>(yi+1, xi);
  float px4 = image.at<float>(yi+1, xi+1);      
  
  // Interpolate pixel intensity.
  float interpolated_value = 
        (1.0-k1)*(1.0-k2)*px1 +
  (     f1     ? ( k1*(1.0-k2)*px2 ) : 0) +
  (     f2     ? ( (1.0-k1)*k2*px3 ) : 0) +            
  ( (f1 && f2) ? ( k1*k2*px4 ) : 0);

  return interpolated_value;
}

float norm(cv::Mat m)
{
    float squareSum = 0.0f;
    
    for(std::size_t row = 0; row < m.rows; ++row)
    {
        for(std::size_t col = 0; col < m.cols; ++col)
        {
            float elem = m.at<float>(static_cast<int>(row), static_cast<int>(col));
            
            squareSum += elem * elem;
        }
    }
    
    return sqrt(squareSum);
}

extern "C"
void inverseCompositional( float* imageArray
                         , float* templateImageArray
                         , float* affineParameterEstimates
                         , const std::size_t imageHeight
                         , const std::size_t imageWidth
                         , const std::size_t templateImageHeight
                         , const std::size_t templateImageWidth
                         , const float epsilon
                         , const int maxIteration
                         )
{
    cv::Mat target = buildImageMat(imageArray, imageHeight, imageWidth);
    cv::Mat templateImageMat = buildImageMat(templateImageArray, templateImageHeight, templateImageWidth);
    
	// Find the 2-D similarity transform that best aligns the two images (uniform scale, rotation and translation)
	cv::Mat debug;

	cv::Mat template_gradient_row;    // Gradient of I in X direction.
	cv::Mat template_gradient_col;    // Gradient of I in Y direction.

									// Here we will store matrices.
	cv::Mat_<float> W(3,3);         // Current value of warp W(x,p)
	cv::Mat_<float> dW(3,3);        // Warp update.
	cv::Mat_<float> idW(3,3);       // Warp update.
	cv::Mat_<float> X(3,1);         // Point in coordinate frame of template.
	cv::Mat_<float> Z(3,1);         // Point in coordinate frame of image.

	cv::Mat_<float> H(6,6);         // Approximate Hessian.
	cv::Mat_<float> b(6,1);         // Vector in the right side of the system of linear equations.
	cv::Mat_<float> delta_p(6,1);   // Parameter update value.

							  // Create images.
	template_gradient_row = cv::Mat(templateImageMat.rows, templateImageMat.cols, CV_32FC1);
	template_gradient_col = cv::Mat(templateImageMat.rows, templateImageMat.cols, CV_32FC1);
    
    float* steepest_descent = new float[6 * templateImageHeight * templateImageWidth];

	//The "magic number" appearing at the end in the following is simply the inverse 
	//of the absolute sum of the weights in the matrix representing the Scharr filter.
	cv::Scharr(templateImageMat, template_gradient_row, -1, 0, 1, 1.0 / 32.0);
	cv::Scharr(templateImageMat, template_gradient_col, -1, 1, 0, 1.0 / 32.0);
    
	H = cv::Mat::zeros(6, 6, CV_32FC1);    
    	
	// Walk through pixels in the template T.
	for(int col = 0; col < templateImageWidth; ++col)
	{
		for(int row = 0; row < templateImageHeight; ++row)
		{
			// Evaluate gradient of T.
			float Tx = template_gradient_col.at<float>(row, col);	
			float Ty = template_gradient_row.at<float>(row, col);	
			
			// Calculate steepest descent image's element.
            steepest_descent[6 * (row * templateImageWidth + col) + 0] = Tx * col;
            steepest_descent[6 * (row * templateImageWidth + col) + 1] = Ty * col;
            steepest_descent[6 * (row * templateImageWidth + col) + 2] = Tx * row;
            steepest_descent[6 * (row * templateImageWidth + col) + 3] = Ty * row;
            steepest_descent[6 * (row * templateImageWidth + col) + 4] = Tx;
            steepest_descent[6 * (row * templateImageWidth + col) + 5] = Ty;
            
			// Add a term to Hessian.
			for(int i = 0; i < 6; ++i)
			{
				for(int j = 0; j < 6; ++j)
				{
                    H.at<float>(i, j) += steepest_descent[6 * (row * templateImageWidth + col) + i] * steepest_descent[6 * (row * templateImageWidth + col) + j];
				}
			}
		}
	}
    
    // Invert Hessian.
    cv::Mat iH = H.inv();

	/*
	 *   Iteration stage.
	 */
     
    buildTransformationFromInput(W, affineParameterEstimates);

	// Here we will store current value of mean error.
	float mean_error=0;

	// Iterate
	int iter=0; // number of current iteration
    while(iter < maxIteration)
	{
		++iter; // Increment iteration counter

        target.copyTo(debug);
        
		mean_error = 0; // Set mean error value with zero

		int pixel_count = 0; // Count of processed pixels
		
		b = cv::Mat::zeros(6, 1, CV_32FC1); // Set b matrix with zeroes
			
		// Walk through pixels in the template T.
		for(int col = 0; col < templateImageWidth; ++col)
		{
			for(int row = 0; row < templateImageHeight; ++row)
			{
				// Set vector X with pixel coordinates (x,y,1)
                X = (cv::Mat_<float>(3,1) << col, row, 1);

				// Warp Z=W*X
                Z = W * X;

				// Get coordinates of warped pixel in coordinate frame of I.
                float col2, row2; // pixel coordinates in the coordinate frame of I.
                col2 = Z.at<float>(0,0);
                row2 = Z.at<float>(1,0);

				// Get the nearest integer pixel coords (x2i;y2i).
				int col2i = int(floor(col2));
				int row2i = int(floor(row2));

				if(col2i >= 0 && col2i < imageWidth && // check if pixel is inside I.
					row2i >= 0 && row2i < imageHeight)
				{
					++pixel_count;

					// Calculate intensity of a transformed pixel with sub-pixel accuracy
					// using bilinear interpolation.
					float I2 = interpolate(target, row2, col2);
                    
                    debug.at<float>(row2i, col2i) = templateImageMat.at<float>(row,col);
                    if(row == 0 || col == 0 || col == templateImageMat.cols-1 || row == templateImageMat.rows-1)
                    {
                        debug.at<float>(row2i,col2i) = 1.0;
                    }

					// Calculate image difference D = I(W(x,p))-T(x).
					float D = I2 - templateImageMat.at<float>(row, col);

					// Update mean error value.
					mean_error += fabs(D);

					// Add a term to b matrix.
					b.at<float>(0,0) += steepest_descent[6 * (row * templateImageWidth + col) + 0] * D;
					b.at<float>(1,0) += steepest_descent[6 * (row * templateImageWidth + col) + 1] * D;
					b.at<float>(2,0) += steepest_descent[6 * (row * templateImageWidth + col) + 2] * D;	
					b.at<float>(3,0) += steepest_descent[6 * (row * templateImageWidth + col) + 3] * D;	
					b.at<float>(4,0) += steepest_descent[6 * (row * templateImageWidth + col) + 4] * D;	
					b.at<float>(5,0) += steepest_descent[6 * (row * templateImageWidth + col) + 5] * D;					
				}
			}
		}

		// Finally, calculate resulting mean error.
		if(pixel_count!=0)
			mean_error /= pixel_count;

		// Find parameter increment.
        delta_p = iH * b;

		init_warp(dW, delta_p);
        
		// Invert warp.
		idW = getWarpInvert(dW);
        
        //W o idW;
		update_warp(W, idW);
		// Print diagnostic information to screen.
		printf("iter=%d mean_error=%f\n", iter, mean_error);
        
        cv::imshow("Debug", debug);
        cv::waitKey(24);

		// Check termination critera.
		if(norm(delta_p) <= epsilon) break;
	}
    
    std::cout << W << std::endl;
    
    cv::imshow("Debug", debug);
    cv::waitKey(0);
}
